#include "hip/hip_runtime.h"
/* This is machine problem 2, part 2: brute force k nearest neighbors
 * You are given a large number of particles, and are asked
 * to find the k particles that are nearest to each one.
 * Look at the example in /tutorials/thread_local_variables.cu
 * for how you can use per thread arrays for sorting.
 * Using that example, port the cpu reference code to the gpu in a first step.
 * In a second step, modify your code so that the per-thread arrays are in 
 * shared memory. You should submit this second version of your code. 
 */
 
/*
 * SUBMISSION INSTRUCTIONS
 * =========================
 * 
 * You can submit the assignment from any of the cluster machines by using
 * our submit script. Th submit script bundles the entire current directory into
 * a submission. Thus, you use it by CDing to a the directory for your assignment,
 * and running:
 * 
 *   > cd *some directory*
 *   > /usr/class/cs193g/bin/submit mp2
 * 
 * This will submit the current directory as your assignment. You can submit
 * as many times as you want, and we will use your last submission.
 */
 
#include <cassert>

#include "mp2-util.h"

// TODO enable this to print debugging information
//const bool print_debug = true;
const bool print_debug = false;

event_pair timer;

inline __device__ __host__ float3 operator -(float3 a, float3 b)
{
  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__host__ __device__
float dist2(float3 a, float3 b)
{
  float3 d = a - b;
  float d2 = d.x*d.x + d.y*d.y + d.z*d.z;
  return d2;
}

template
<typename T>
__host__ __device__
void init_list(T *base_ptr, unsigned int size, T val)
{
  for(int i=0;i<size;i++)
  {
    base_ptr[i] = val;
  }
}

__host__ __device__
void insert_list(float *dist_list, int *id_list, int size, float dist, int id)
{
 int k;
 for (k=0; k < size; k++) {
   if (dist < dist_list[k]) {
     // we should insert it in here, so push back and make it happen
     for (int j = size - 1; j > k ; j--) {
       dist_list[j] = dist_list[j-1];
       id_list[j] = id_list[j-1];
     }
     dist_list[k] = dist;
     id_list[k] = id;
     break;
   }
 }
}

template
  <int num_neighbors>
void host_find_knn(float3 *particles, int *knn, int array_length)
{
  for(int i=0;i<array_length;i++)
  {
    float3 p = particles[i];
    float neigh_dist[num_neighbors];
    int neigh_ids[num_neighbors];
    
    init_list(&neigh_dist[0],num_neighbors,2.0f);
    init_list(&neigh_ids[0],num_neighbors,-1);
    for(int j=0;j<array_length;j++)
    {
      if(i != j)
      {
        float rsq = dist2(p,particles[j]);
        insert_list(&neigh_dist[0], &neigh_ids[0], num_neighbors, rsq, j);
      }
    }
    for(int j=0;j<num_neighbors;j++)
    {
      knn[num_neighbors*i + j] = neigh_ids[j];
    }
  }
}

template
  <int num_neighbors>
__global__ void device_find_knn_local_mem(float3 *particles, int *knn, int array_length)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= array_length) return;

  {
    float3 p = particles[i];
    float neigh_dist[num_neighbors];
    int neigh_ids[num_neighbors];

    init_list(&neigh_dist[0],num_neighbors,2.0f);
    init_list(&neigh_ids[0],num_neighbors,-1);
    for(int j=0;j<array_length;j++)
    {
      if(i != j)
      {
        float rsq = dist2(p,particles[j]);
        insert_list(&neigh_dist[0], &neigh_ids[0], num_neighbors, rsq, j);
      }
    }
    for(int j=0;j<num_neighbors;j++)
    {
      knn[num_neighbors*i + j] = neigh_ids[j];
    }
  }
}

template
  <int num_neighbors>
__global__ void device_find_knn_shared_mem(float3 *particles, int *knn, int array_length)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= array_length) return;

  {
    float3 p = particles[i];
    extern __shared__ float neigh_dist[]; 
                      int * neigh_ids = (int*)&neigh_dist[num_neighbors];

    init_list(&neigh_dist[0],num_neighbors,2.0f);
    init_list(&neigh_ids[0],num_neighbors,-1);
    for(int j=0;j<array_length;j++)
    {
      if(i != j)
      {
        float rsq = dist2(p,particles[j]);
        insert_list(&neigh_dist[0], &neigh_ids[0], num_neighbors, rsq, j);
      }
    }
    for(int j=0;j<num_neighbors;j++)
    {
      knn[num_neighbors*i + j] = neigh_ids[j];
    }
  }
}


void allocate_host_memory(int num_particles, int num_neighbors,
                          float3 *&h_particles, int *&h_knn, int *&h_knn_checker)
{
  // malloc host array
  h_particles = (float3*)malloc(num_particles * sizeof(float3));
  h_knn = (int*)malloc(num_particles * num_neighbors * sizeof(int));
  h_knn_checker = (int*)malloc(num_particles * num_neighbors * sizeof(int));

  // if either memory allocation failed, report an error message
  if(h_particles == 0 || h_knn == 0 || h_knn_checker == 0)
  {
    printf("couldn't allocate host memory\n");
    exit(1);
  }
}

#define CHECK(call) { \
  hipError_t err = hipSuccess; \
  if ( (err = (call)) != hipSuccess) { \
    fprintf(stderr, "Got error %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
    exit(1); \
  }\
}

void allocate_device_memory(int num_particles, int num_neighbors,
                            float3 *&d_particles, int *&d_knn)
{
  // device memory allocations here
  CHECK(hipMalloc((void**)&d_particles, num_particles * sizeof(float3)));
  CHECK(hipMalloc((void**)&d_knn, num_particles * num_neighbors *  sizeof(int)));
}


void deallocate_host_memory(float3 *h_particles, int *h_knn, int *h_knn_checker)
{
  free(h_particles);
  free(h_knn);
  free(h_knn_checker);
}


void deallocate_device_memory(float3 *d_particles, int *d_knn)
{
  // device memory deallocations here
  CHECK(hipFree(d_particles));
  CHECK(hipFree(d_knn));
}


bool cross_check_results(int * reference_knn, int * knn, int num_particles, int num_neighbors)
{
  int error = 0;


  for(int i=0;i<num_particles;i++)
  {
    for(int j=0;j<num_neighbors;j++)
    {
      if(reference_knn[i*num_neighbors + j] != knn[i*num_neighbors + j])
      {
        if(print_debug) printf("particle %d, neighbor %d is %d on cpu, %d on gpu\n",i,j,reference_knn[i*num_neighbors + j],knn[i*num_neighbors + j]);
        error = 1;
      }
    }

  }

  if(error)
  {
    printf("Output of CUDA version and normal version didn't match! \n");
  }
  else {
    printf("Worked! CUDA and reference output match. \n");
  }
  return error;
}

int main(void)
{  
  // create arrays of 8K elements
  int num_particles = 20*1024;
  const int num_neighbors = 5;

  // pointers to host arrays
  float3 *h_particles = 0;
  int    *h_knn = 0;
  int    *h_knn_checker = 0;

  // pointers to device arrays
  float3 *d_particles = 0;
  int    *d_knn = 0;

  allocate_host_memory(num_particles, num_neighbors, h_particles, h_knn, h_knn_checker);
  allocate_device_memory(num_particles, num_neighbors, d_particles, d_knn);

  // generate random input
  // initialize
  srand(13);

  for(int i=0;i< num_particles;i++)
  {
    h_particles[i] = make_float3((float)rand()/(float)RAND_MAX,(float)rand()/(float)RAND_MAX,(float)rand()/(float)RAND_MAX);
  }

  // copy input to GPU
  start_timer(&timer);
  //copy of input from host to device here
  CHECK(hipMemcpy(d_particles,h_particles, num_particles * sizeof (float3), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_knn,h_knn, num_particles * num_neighbors * sizeof (int), hipMemcpyHostToDevice));
  stop_timer(&timer,"copy to gpu");

  dim3 THRD_SZ(512);
  dim3 GRID_SZ((num_particles + THRD_SZ.x-1)/THRD_SZ.x);

  start_timer(&timer);  
  // kernel launch which uses local memory
  device_find_knn_local_mem<num_neighbors><<<GRID_SZ, THRD_SZ>>>(d_particles, d_knn, num_particles);
  check_cuda_error("brute force knn");
  stop_timer(&timer,"brute force knn");

  start_timer(&timer);  
  // kernel launch which uses __shared__ memory
  int shmem_size = THRD_SZ.x * (sizeof (float) +  sizeof(int)) * num_neighbors;
  device_find_knn_shared_mem<num_neighbors><<<GRID_SZ, shmem_size>>>(d_particles, d_knn, num_particles);
  check_cuda_error("shared meme knn");
  stop_timer(&timer,"shared mem knn");

  // download and inspect the result on the host
  start_timer(&timer);
  // copy results from device to host here
  hipMemcpy(h_knn,d_knn, num_particles * num_neighbors * sizeof (int), hipMemcpyDeviceToHost);
  check_cuda_error("copy from gpu");
  stop_timer(&timer,"copy back from gpu memory");

  // generate reference output
  start_timer(&timer);
  host_find_knn<num_neighbors>(h_particles, h_knn_checker, num_particles);
  stop_timer(&timer,"cpu brute force knn");

  // check CUDA output versus reference output
  cross_check_results(h_knn_checker, h_knn, num_particles, num_neighbors);

  deallocate_host_memory(h_particles, h_knn, h_knn_checker);
  deallocate_device_memory(d_particles, d_knn);

  return 0;
}

